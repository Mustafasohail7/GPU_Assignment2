#include "hip/hip_runtime.h"
#include <iostream>
#include <Eigen/Dense>
#include <hip/hip_runtime.h>

__global__ void DeviceMatrixMultiply(float *M, float *N, float *P, int rows,
                                    int cols, int common)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < rows) && (col < cols))
    {
        float Pvalue = 0;
        // each thread computes one element of the block sub-matrix
        for (int k = 0; k < common; ++k)
        {
            Pvalue += M[k * rows + row] * N[col * common + k];
        }
        P[col * rows + row] = Pvalue;
    }
}

__global__ void DeviceMatrixScalarMultiply(float *M, float N, float *P, int rows, int cols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < rows) && (col < cols))
    {
        P[col * rows + row] = M[col * rows + row] * N;
    }
}

Eigen::MatrixXf HostMatrixMultiply(const Eigen::MatrixXf &M, const Eigen::MatrixXf &N)
{
    int rows = M.rows();
    int cols = N.cols();
    if (M.cols() != N.rows())
    {
        std::cout << M.rows() << "," << M.cols() << std::endl;
        std::cout << N.rows() << "," << N.cols() << std::endl;
        std::cout << "Matrix dimensions are not compatible for multiplication" << std::endl;
        return Eigen::MatrixXf::Zero(1, 1);
    }
    int common = M.cols();
    float *d_M, *d_N, *d_P;
    int size_M = rows * common * sizeof(float);
    int size_N = common * cols * sizeof(float);
    int size_P = rows * cols * sizeof(float);

    hipMalloc((void **)&d_M, size_M);
    hipMalloc((void **)&d_N, size_N);
    hipMalloc((void **)&d_P, size_P);
    hipMemcpy(d_M, M.data(), size_M, hipMemcpyHostToDevice);
    hipMemcpy(d_N, N.data(), size_N, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x, (cols + dimBlock.y - 1) / dimBlock.y);
    DeviceMatrixMultiply<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, rows, cols, common);
    hipDeviceSynchronize();

    Eigen::MatrixXf P(rows, cols);
    checkCudaErr(hipMemcpy(P.data(), d_P, size_P, hipMemcpyDeviceToHost), "Memcpy P");
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return P;
}

Eigen::MatrixXf HostMatrixScalarMultiply(const Eigen::MatrixXf &M, float N)
{
    int rows = M.rows();
    int cols = M.cols();
    float *d_M, *d_P;
    int size_M = rows * cols * sizeof(float);
    int size_P = rows * cols * sizeof(float);
    hipMalloc((void **)&d_M, size_M);
    hipMalloc((void **)&d_P, size_P);
    hipMemcpy(d_M, M.data(), size_M, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x, (cols + dimBlock.y - 1) / dimBlock.y);
    DeviceMatrixScalarMultiply<<<dimGrid, dimBlock>>>(d_M, N, d_P, rows, cols);
    hipDeviceSynchronize();

    Eigen::MatrixXf P(rows, cols);
    hipMemcpy(P.data(), d_P, size_P, hipMemcpyDeviceToHost);
    hipFree(d_M);
    hipFree(d_P);

    return P;
}

__global__ void DeviceMatrixAddition(float *M, float *N, float *P, int rows, int cols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < rows) && (col < cols))
    {
        P[col * rows + row] = M[col * rows + row] + N[col * rows + row];
    }
}

Eigen::MatrixXf HostMatrixAddition(const Eigen::MatrixXf &M, const Eigen::MatrixXf &N)
{
    int rows = M.rows();
    int cols = M.cols();
    float *d_M, *d_N, *d_P;
    int size_M = rows * cols * sizeof(float);
    int size_P = rows * cols * sizeof(float);
    hipMalloc((void **)&d_M, size_M);
    hipMalloc((void **)&d_N, size_M);
    hipMalloc((void **)&d_P, size_P);
    hipMemcpy(d_M, M.data(), size_M, hipMemcpyHostToDevice);
    hipMemcpy(d_N, N.data(), size_M, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x, (cols + dimBlock.y - 1) / dimBlock.y);
    DeviceMatrixAddition<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, rows, cols);
    hipDeviceSynchronize();

    Eigen::MatrixXf P(rows, cols);
    hipMemcpy(P.data(), d_P, size_P, hipMemcpyDeviceToHost);
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return P;
}

__global__ void DeviceMatrixSubtraction(float *M, float *N, float *P, int rows, int cols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if ((row < rows) && (col < cols))
    {
        P[col * rows + row] = M[col * rows + row] - N[col * rows + row];
    }
}

Eigen::MatrixXf HostMatrixSubtraction(const Eigen::MatrixXf &M, const Eigen::MatrixXf &N)
{
    int rows = M.rows();
    int cols = M.cols();
    float *d_M, *d_N, *d_P;
    int size_M = rows * cols * sizeof(float);
    int size_P = rows * cols * sizeof(float);
    hipMalloc((void **)&d_M, size_M);
    hipMalloc((void **)&d_N, size_M);
    hipMalloc((void **)&d_P, size_P);
    hipMemcpy(d_M, M.data(), size_M, hipMemcpyHostToDevice);
    hipMemcpy(d_N, N.data(), size_M, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x, (cols + dimBlock.y - 1) / dimBlock.y);
    DeviceMatrixSubtraction<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, rows, cols);
    hipDeviceSynchronize();


    Eigen::MatrixXf P(rows, cols);
    hipMemcpy(P.data(), d_P, size_P, hipMemcpyDeviceToHost);
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return P;
}