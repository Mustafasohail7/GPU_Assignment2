#include "hip/hip_runtime.h"
#include <iostream>
#include <Eigen/Dense>
#include <hip/hip_runtime.h>

__global__ void MatrixScalarMultiply(float *M, float N, float *P, int rows, int cols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < rows && col < cols)
    {
        float result = M[col * rows + row] * N;
        P[col * rows + row] = result;
    }
}

__global__ void DeviceMatrixMultiply(float *M, float *N, float *P, int rows,
                                    int cols, int common)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < rows && col < cols)
    {
        float Pvalue = 0;

        for (int k = 0; k < common; ++k)
        {
            float Mvalue = M[k * rows + row];
            float Nvalue = N[col * common + k];
            Pvalue += Mvalue * Nvalue;
        }

        P[col * rows + row] = Pvalue;
    }
}

__global__ void DeviceMatrixAddition(float *M, float *N, float *P, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < cols && row < rows)
    {
        int index = col * rows + row;
        P[index] = M[index] + N[index];
    }
}

__global__ void DeviceMatrixSubtraction(float *M, float *N, float *P, int rows, int cols)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < cols && row < rows)
    {
        int index = col * rows + row;
        P[index] = M[index] - N[index];
    }
}

Eigen::MatrixXf HostMatrixMultiply(const Eigen::MatrixXf &M, const Eigen::MatrixXf &N)
{
    int rows = M.rows();
    int cols = N.cols();
    int common = M.cols();

    float *d_M, *d_N, *d_P;
    int size_M = rows * common * sizeof(float);
    int size_N = common * cols * sizeof(float);
    int size_P = rows * cols * sizeof(float);

    hipMalloc((void **)&d_M, size_M);
    hipMalloc((void **)&d_N, size_N);
    hipMalloc((void **)&d_P, size_P);

    hipMemcpy(d_M, M.data(), size_M, hipMemcpyHostToDevice);
    hipMemcpy(d_N, N.data(), size_N, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x, (cols + dimBlock.y - 1) / dimBlock.y);

    // DeviceMatrixMultiply kernel
    {
        DeviceMatrixMultiply<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, rows, cols, common);
        hipDeviceSynchronize();
    }

    Eigen::MatrixXf P(rows, cols);
    hipMemcpy(P.data(), d_P, size_P, hipMemcpyDeviceToHost);

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return P;
}

Eigen::MatrixXf HostMatrixScalarMultiply(const Eigen::MatrixXf &M, float N)
{
    int rows = M.rows();
    int cols = M.cols();

    float *d_M, *d_P;

    int size_M = rows * cols * sizeof(float);
    int size_P = rows * cols * sizeof(float);

    hipMalloc((void **)&d_M, size_M);
    hipMalloc((void **)&d_P, size_P);

    hipMemcpy(d_M, M.data(), size_M, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x, (cols + dimBlock.y - 1) / dimBlock.y);

    DeviceMatrixScalarMultiply<<<dimGrid, dimBlock>>>(d_M, N, d_P, rows, cols);

    hipDeviceSynchronize();

    Eigen::MatrixXf P(rows, cols);
    
    hipMemcpy(P.data(), d_P, size_P, hipMemcpyDeviceToHost);
    hipFree(d_M);
    hipFree(d_P);

    return P;
}
Eigen::MatrixXf ScalarMultiplyMatrix(const Eigen::MatrixXf &matrix, float scalar)
{
    int rows = matrix.rows();
    int cols = matrix.cols();

    float *deviceMatrix, *deviceResult;

    int sizeMatrix = rows * cols * sizeof(float);
    int sizeResult = rows * cols * sizeof(float);

    hipMalloc((void **)&deviceMatrix, sizeMatrix);
    hipMalloc((void **)&deviceResult, sizeResult);

    hipMemcpy(deviceMatrix, matrix.data(), sizeMatrix, hipMemcpyHostToDevice);

    dim3 blockDims(16, 16);
    dim3 gridDims((rows + blockDims.x - 1) / blockDims.x, (cols + blockDims.y - 1) / blockDims.y);

    ScalarMultiplyKernel<<<gridDims, blockDims>>>(deviceMatrix, scalar, deviceResult, rows, cols);

    hipDeviceSynchronize();

    Eigen::MatrixXf result(rows, cols);
    
    hipMemcpy(result.data(), deviceResult, sizeResult, hipMemcpyDeviceToHost);
    hipFree(deviceMatrix);
    hipFree(deviceResult);

    return result;
}


Eigen::MatrixXf HostMatrixAddition(const Eigen::MatrixXf &M, const Eigen::MatrixXf &N)
{
    int rows = M.rows();
    int cols = M.cols();

    float *deviceM, *deviceN, *deviceP;

    int sizeM = rows * cols * sizeof(float);
    int sizeP = rows * cols * sizeof(float);

    hipMalloc((void **)&deviceM, sizeM);
    hipMalloc((void **)&deviceN, sizeM);
    hipMalloc((void **)&deviceP, sizeP);

    hipMemcpy(deviceM, M.data(), sizeM, hipMemcpyHostToDevice);
    hipMemcpy(deviceN, N.data(), sizeM, hipMemcpyHostToDevice);

    dim3 blockDims(16, 16);
    dim3 gridDims((rows + blockDims.x - 1) / blockDims.x, (cols + blockDims.y - 1) / blockDims.y);

    DeviceMatrixAddition<<<gridDims, blockDims>>>(deviceM, deviceN, deviceP, rows, cols);

    hipDeviceSynchronize();

    Eigen::MatrixXf P(rows, cols);
    
    hipMemcpy(P.data(), deviceP, sizeP, hipMemcpyDeviceToHost);
    hipFree(deviceM);
    hipFree(deviceN);
    hipFree(deviceP);

    return P;
}



Eigen::MatrixXf HostMatrixSubtraction(const Eigen::MatrixXf &M, const Eigen::MatrixXf &N)
{
    int rows = M.rows();
    int cols = M.cols();

    float *deviceM, *deviceN, *deviceP;

    int sizeM = rows * cols * sizeof(float);
    int sizeP = rows * cols * sizeof(float);

    hipMalloc((void **)&deviceM, sizeM);
    hipMalloc((void **)&deviceN, sizeM);
    hipMalloc((void **)&deviceP, sizeP);

    hipMemcpy(deviceM, M.data(), sizeM, hipMemcpyHostToDevice);
    hipMemcpy(deviceN, N.data(), sizeM, hipMemcpyHostToDevice);

    dim3 blockDims(16, 16);
    dim3 gridDims((rows + blockDims.x - 1) / blockDims.x, (cols + blockDims.y - 1) / blockDims.y);

    MatrixSubtractionKernel<<<gridDims, blockDims>>>(deviceM, deviceN, deviceP, rows, cols);

    hipDeviceSynchronize();

    Eigen::MatrixXf P(rows, cols);
    hipMemcpy(P.data(), deviceP, sizeP, hipMemcpyDeviceToHost);
    hipFree(deviceM);
    hipFree(deviceN);
    hipFree(deviceP);

    return P;
}